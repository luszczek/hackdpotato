
#include <cstdlib>

#include <iostream>

#include <hip/hip_runtime.h>

__global__ void increment(float *val)
{
  val[0]++;
}

void
run_par_gpu() {
  int device;
  const int ngpu = 2;
  float *values[ngpu], currentDevice, *fromDevice;

  fromDevice = (float *)malloc(ngpu * sizeof(float));

  for (device = 0; device < ngpu; device++) {
    hipSetDevice(device);

    hipMalloc((void **)&(values[device]), 1 * sizeof(float));

    currentDevice = device;

    hipMemcpy(values[device], &currentDevice, 1*sizeof(float), hipMemcpyHostToDevice);

    increment<<<1,1>>>(values[device]);

    hipMemcpy(&fromDevice[device], values[device], 1*sizeof(float), hipMemcpyDeviceToHost);
  }

  for (device = 0; device < ngpu; device++) {
    std::cout << fromDevice[device] << std::endl;
  }
}

int
main(void) {
  run_par_gpu();
  return 0;
}
