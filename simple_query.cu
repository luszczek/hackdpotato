
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <mpi.h>

int WorldRank, WorldSize;

void
query_one_device(int device) {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);
  printf("device[%d].major %d.\n", device, deviceProp.major);
  printf("device[%d].minor %d.\n", device, deviceProp.minor);
}

void
query_all() {
  int deviceCount;

  hipGetDeviceCount(&deviceCount);

  for (int device=0; device < deviceCount; ++device)
    query_one_device(device);
}

void
random_generate() {
  hiprandGenerator_t gen;
  int seed=1;

  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
  hiprandSetPseudoRandomGeneratorSeed(gen, seed);
  hiprandGenerateUniformDouble(gen, NULL, 1);
  hiprandDestroyGenerator(gen);
}

int
main(int argc, char *argv[]) {
  MPI_Init(&argc, &argv);

  MPI_Comm_size(MPI_COMM_WORLD, &WorldSize);
  MPI_Comm_rank(MPI_COMM_WORLD, &WorldRank);

  for (int rank = 0; rank < WorldSize; ++rank) {
    MPI_Barrier(MPI_COMM_WORLD);

    if (rank == WorldRank) {
      printf("%d/%d: ", WorldRank, WorldSize );
      query_all();
      fflush(stdout);
    }
  }

  MPI_Finalize();
  return 0;
}
